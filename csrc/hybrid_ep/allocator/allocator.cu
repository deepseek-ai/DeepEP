#include "allocator.cuh"

// Check if the current device supports fabric.
bool ExtendedMemoryAllocator::support_fabric() {
  int device_count;
  CUDA_CHECK(hipGetDeviceCount(&device_count));

  for (int device = 0; device < device_count; ++device) {
    int support = 0;
    CU_CHECK(
        hipDeviceGetAttribute(&support, CU_DEVICE_ATTRIBUTE_HANDLE_TYPE_FABRIC_SUPPORTED, device));
    if (!support) {
      return false;
    }
  }
  return true;
}

// Round-up allocation size to fabric granularity.
size_t inline get_size_align_to_granularity(size_t size_raw, size_t granularity) {
  size_t size = (size_raw + granularity - 1) & ~(granularity - 1);
  if (size == 0)
    size = granularity;
  return size;
}

void ExtendedMemoryAllocator::init(bool enable_fabric) {
  this->support_fabric_ = this->support_fabric();
  enable_fabric_ = enable_fabric;

  if (support_fabric_ && enable_fabric_) {
    int device_id = -1;
    // It seems a dummy call to set the device. but it is useful to prevent the invalid device context error in gb..
    CUDA_CHECK(hipGetDevice(&device_id));
    CUDA_CHECK(hipSetDevice(device_id));
    // Get the device context.
    CU_CHECK(hipCtxGetDevice(&device_));
    fabric_prop_.type = hipMemAllocationTypePinned;
    fabric_prop_.location.type = hipMemLocationTypeDevice;
    fabric_prop_.requestedHandleTypes = CU_MEM_HANDLE_TYPE_FABRIC;
    fabric_prop_.location.id = device_;
    CU_CHECK(hipMemGetAllocationGranularity(&fabric_granularity_, &fabric_prop_,
                                           hipMemAllocationGranularityMinimum));
    access_desc.location.type = hipMemLocationTypeDevice;
    access_desc.location.id = device_;
    access_desc.flags = hipMemAccessFlagsProtReadWrite;
  }
  if (!support_fabric_) {
    enable_fabric_ = false;
  }
}

void ExtendedMemoryAllocator::allocate(void** ptr, size_t size_raw) {
  if (enable_fabric_) {
    size_t size = get_size_align_to_granularity(size_raw, fabric_granularity_);
    hipMemGenericAllocationHandle_t handle;
    CU_CHECK(hipMemCreate(&handle, size, &fabric_prop_, 0));
    CU_CHECK(hipMemAddressReserve((hipDeviceptr_t*)ptr, size, fabric_granularity_, 0, 0));
    CU_CHECK(hipMemMap((hipDeviceptr_t)*ptr, size, 0, handle, 0));
    CU_CHECK(hipMemSetAccess((hipDeviceptr_t)*ptr, size, &access_desc, 1));
  } else {
    CUDA_CHECK(hipMalloc(ptr, size_raw));
  }
}

void ExtendedMemoryAllocator::free(void* ptr) {
  if (enable_fabric_) {
    hipMemGenericAllocationHandle_t handle;
    CU_CHECK(hipMemRetainAllocationHandle(&handle, ptr));
    size_t size = 0;
    CU_CHECK(hipMemGetAddressRange(NULL, &size, (hipDeviceptr_t)ptr));
    CU_CHECK(hipMemUnmap((hipDeviceptr_t)ptr, size));
    CU_CHECK(hipMemAddressFree((hipDeviceptr_t)ptr, size));
    CU_CHECK(hipMemRelease(handle));
  } else {
    CUDA_CHECK(hipFree(ptr));
  }
}

void ExtendedMemoryAllocator::get_handle(MemHandle* mem_handle, void* ptr) {
  size_t size = 0;
  CU_CHECK(hipMemGetAddressRange(NULL, &size, (hipDeviceptr_t)ptr));
  
  mem_handle->size = size;
  if (enable_fabric_) {
    hipMemGenericAllocationHandle_t handle;
    CU_CHECK(hipMemRetainAllocationHandle(&handle, ptr));
    CU_CHECK(hipMemExportToShareableHandle(&mem_handle->inner.cu_mem_fabric_handle, handle,
                                          CU_MEM_HANDLE_TYPE_FABRIC, 0));
  } else {
    CUDA_CHECK(hipIpcGetMemHandle(&mem_handle->inner.cuda_ipc_mem_handle, ptr));
  }
}

void ExtendedMemoryAllocator::open_handle(void** ptr, MemHandle* mem_handle) {
  if (enable_fabric_) {
    size_t size = mem_handle->size;
    hipMemGenericAllocationHandle_t handle;
    CU_CHECK(hipMemImportFromShareableHandle(&handle, &mem_handle->inner.cu_mem_fabric_handle,
                                            CU_MEM_HANDLE_TYPE_FABRIC));
    CU_CHECK(hipMemAddressReserve((hipDeviceptr_t*)ptr, size, 0, 0, 0));
    CU_CHECK(hipMemMap((hipDeviceptr_t)*ptr, size, 0, handle, 0));
    CU_CHECK(hipMemSetAccess((hipDeviceptr_t)*ptr, size, &access_desc, 1));
  } else {
    CUDA_CHECK(hipIpcOpenMemHandle(ptr, mem_handle->inner.cuda_ipc_mem_handle,
                                    hipIpcMemLazyEnablePeerAccess));
  }
}

void ExtendedMemoryAllocator::close_handle(void* ptr) {
  if (enable_fabric_) {
    size_t size = 0;
    CU_CHECK(hipMemGetAddressRange(NULL, &size, (hipDeviceptr_t)ptr));
    CU_CHECK(hipMemUnmap((hipDeviceptr_t)ptr, size));
    CU_CHECK(hipMemAddressFree((hipDeviceptr_t)ptr, size));
  } else {
    CUDA_CHECK(hipIpcCloseMemHandle(ptr));
  }
}
